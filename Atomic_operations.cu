#include "hip/hip_runtime.h"
// By: Krishna Pal Deora , 18JE0425. , Integrated M.Tech. Mathematics & Computing (6th semester)


#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>

/**
 * This example illustrates implementation of custom atomic operations using
 * CUDA's built-in atomicCAS function to implement atomic signed 32-bit integer
 * atomicAdd,atomicSub,atomicMax,atomicMin.
 **/

 // atomicAdd()
 
__device__ int myAtomicAdd(int *address, int incr)
{
    // Create an initial guess for the value stored at *address.
    int guess = *address;
    int oldValue = atomicCAS(address, guess, guess + incr);

    // Loop while the guess is incorrect.
    while (oldValue != guess)
    {
        guess = oldValue;
        oldValue = atomicCAS(address, guess, guess + incr);
    }

    return oldValue;
}

//atomicSub()

__device__ int atomicSub(int* address, int val);
unsigned int atomicSub(unsigned int* address,unsigned int val);

 atomicMax()

__device__ int atomicMax(int* address, int val);
unsigned int atomicMax(unsigned int* address,unsigned int val);
unsigned long long int atomicMax(unsigned long long int* address,unsigned long long int val);

// atomicMin()
__device__ int atomicMin(int* address, int val);
unsigned int atomicMin(unsigned int* address,unsigned int val);
unsigned long long int atomicMin(unsigned long long int* address,unsigned long long int val);



__global__ void kernel(int *sharedInteger)
{
    myAtomicAdd(sharedInteger, 1);
}

int main(int argc, char **argv)
{
    int h_sharedInteger;
    int *d_sharedInteger;
    CHECK(hipMalloc((void **)&d_sharedInteger, sizeof(int)));
    CHECK(hipMemset(d_sharedInteger, 0x00, sizeof(int)));

    kernel<<<4, 128>>>(d_sharedInteger);

    CHECK(hipMemcpy(&h_sharedInteger, d_sharedInteger, sizeof(int),
                     hipMemcpyDeviceToHost));
    printf("4 x 128 increments led to value of %d\n", h_sharedInteger);

    return 0;
}

